#include "hip/hip_runtime.h"
#include "utils.h"
#include "image.h"

#define TMATXB_PARAMS_GEN                                              \
        const int*restrict ipole, const int*restrict pglob, const int*restrict ploc, const int*restrict ieblst, const int*restrict eblst   \
        , const real*restrict x, const real*restrict y, const real*restrict z, const real*restrict pdamp, const real*restrict thole, const real*restrict polarity   \
        , const real (*restrict mu)[6] , real (*restrict efi)[6]                                               \
        , const int npolelocnlb, const int npolelocnlb_pair, const int npolebloc, const int n, const int nproc, const int balanced \
        , const real cut2, const real alsq2, const real alsq2n, const real aewald
#define TMATXB_PARAMS                                               \
        TMATXB_PARAMS_GEN , MIDPOINTIMAGE_PARAMS
#define TMATXB_PARAMS1                                              \
        TMATXB_PARAMS_GEN , MIDPOINTIMAGE1_PARAMS

#define EFLD0_PARAMS_GEN                                              \
        const int*restrict ipole, const int*restrict pglob, const int*restrict ploc, const int*restrict ieblst, const int*restrict eblst   \
        , const real*restrict x, const real*restrict y, const real*restrict z, const real*restrict pdamp, const real*restrict thole, const real*restrict polarity, const real (*restrict rpole)[13]  \
        , real (*restrict efi)[6]                                               \
        , const int npolelocnlb, const int npolelocnlb_pair, const int npolebloc, const int n, const int nproc, const int balanced \
        , const real cut2, const real alsq2, const real alsq2n, const real aewald
#define OTFDC_EFLD0_PARAMS_GEN                                              \
        const int*restrict ipole, const int*restrict pglob, const int*restrict ploc, const int*restrict grplst, const int*restrict atmofst, const int*restrict npergrp, const int*restrict kofst, const int*restrict ieblst, const int*restrict eblst   \
        , const real*restrict x, const real*restrict y, const real*restrict z, const real*restrict pdamp, const real*restrict thole, const real*restrict polarity, const real (*restrict rpole)[13]  \
        , real (*restrict efi)[6], real* restrict zmat                            \
        , const int npolelocnlb, const int npolelocnlb_pair, const int npolebloc, const int n, const int nproc \
        , const real cut2, const real alsq2, const real alsq2n, const real aewald

#define EFLD0_PARAMS                                               \
        EFLD0_PARAMS_GEN , MIDPOINTIMAGE_PARAMS
#define OTFDC_EFLD0_PARAMS                                         \
        OTFDC_EFLD0_PARAMS_GEN , MIDPOINTIMAGE_PARAMS
#define EFLD0_PARAMS1                                              \
        EFLD0_PARAMS_GEN , MIDPOINTIMAGE1_PARAMS
#define OTFDC_EFLD0_PARAMS1                                        \
        OTFDC_EFLD0_PARAMS_GEN , MIDPOINTIMAGE1_PARAMS

#define TMATXB_ARGS                                                 \
          ipole,pglob,ploc,ieblst,eblst,x,y,z,pdamp,thole,polarity  \
        , mu,efi                                                    \
        , npolelocnlb,npolelocnlb_pair,npolebloc,n,nproc,balanced   \
        , cut2,alsq2,alsq2n,aewald                                  \
        , MIDPOINTIMAGE_ARGS

#define EFLD0_ARGS                                                       \
          ipole,pglob,ploc,ieblst,eblst,x,y,z,pdamp,thole,polarity,rpole \
        , efi                                                            \
        , npolelocnlb,npolelocnlb_pair,npolebloc,n,nproc,balanced        \
        , cut2,alsq2,alsq2n,aewald                                       \
        , MIDPOINTIMAGE_ARGS
#define OTFDC_EFLD0_ARGS                                                 \
          ipole,pglob,ploc,grplst,atmofst,npergrp,kofst,ieblst,eblst     \
        , x,y,z,pdamp,thole,polarity,rpole,efi,zmat                      \
        , npolelocnlb,npolelocnlb_pair,npolebloc,n,nproc                 \
        , cut2,alsq2,alsq2n,aewald                                       \
        , MIDPOINTIMAGE_ARGS

__device__ inline
void tmatxb_couple
        ( real& d2, const real3& dist, const real6& dpui, const real6& dpuk, const real& sdamp, const real& pgamma, const real& aewald, const real& alsq2, const real& alsq2n, const real& uscale,
                       real3& fid, real3& fip, real3& fkd, real3& fkp ){
   real d1,ralpha,exp2a;
   real bn0,bn1,bn2;
   real sdamp1,expdamp1;
   real rr3,rr5,rr3_bn1,rr5_bn2,sc3,sc5;
   real duir,dukr,puir,pukr;  /* Scalar products duir = (du(i).r) */

   /* compute the distances and the scaling factors according to Thole's model. */
   d1      = f_sqrt(d2);
   d2      = 1/d2;

   ralpha  = aewald * d1;
   d1      = 1/d1;
   exp2a   = f_exp(-ralpha*ralpha);
   bn0     = f_erfc(ralpha);  // becomes a macro with USE_ERFC_HASTINGS
   bn0     = bn0 * d1;
   bn1     = (     bn0 +         alsq2 * alsq2n * exp2a) * d2;
   bn2     = ( 3 * bn1 + alsq2 * alsq2 * alsq2n * exp2a) * d2;

   if( sdamp == 0.0 ) {
     sdamp1  = -100.0;
     sc3      =   1 - f_exp(sdamp1) * uscale;
     sc5      =   1 - f_exp(sdamp1) * uscale * (1 - sdamp1);
   }
   else {
     sdamp1 = - pgamma / ((d1*sdamp)*(d1*sdamp)*(d1*sdamp));
     if (sdamp1 > -50.0) {
       expdamp1 = f_exp(sdamp1);
       sc3      =   1 - expdamp1 * uscale;
       sc5      =   1 - expdamp1 * uscale * (1 - sdamp1);
     }
     else {
       sc3     = 1;
       sc5     = 1;
     }
   }

   /* compute the field */
   rr3     =     (1 - sc3) * (d1 * d2);
   rr5     = 3 * (1 - sc5) * (d1 * d2 * d2);
   rr3_bn1 = rr3 - bn1;
   rr5_bn2 = rr5 - bn2;

   duir    = dpui.x *dist.x + dpui.y *dist.y + dpui.z *dist.z;
   dukr    = dpuk.x *dist.x + dpuk.y *dist.y + dpuk.z *dist.z;

   puir    = dpui.xx *dist.x + dpui.yy *dist.y + dpui.zz *dist.z;
   pukr    = dpuk.xx *dist.x + dpuk.yy *dist.y + dpuk.zz *dist.z;

   fid.x  += -rr3_bn1*dpuk.x  + rr5_bn2*dukr*dist.x;
   fid.y  += -rr3_bn1*dpuk.y  + rr5_bn2*dukr*dist.y;
   fid.z  += -rr3_bn1*dpuk.z  + rr5_bn2*dukr*dist.z;

   fip.x  += -rr3_bn1*dpuk.xx + rr5_bn2*pukr*dist.x;
   fip.y  += -rr3_bn1*dpuk.yy + rr5_bn2*pukr*dist.y;
   fip.z  += -rr3_bn1*dpuk.zz + rr5_bn2*pukr*dist.z;

   fkd.x  += -rr3_bn1*dpui.x  + rr5_bn2*duir*dist.x;
   fkd.y  += -rr3_bn1*dpui.y  + rr5_bn2*duir*dist.y;
   fkd.z  += -rr3_bn1*dpui.z  + rr5_bn2*duir*dist.z;

   fkp.x  += -rr3_bn1*dpui.xx + rr5_bn2*puir*dist.x;
   fkp.y  += -rr3_bn1*dpui.yy + rr5_bn2*puir*dist.y;
   fkp.z  += -rr3_bn1*dpui.zz + rr5_bn2*puir*dist.z;
}

__device__ inline
void efld0_couple(const real d2,const real3& pos, const rpole_elt& ip, const rpole_elt& kp, const real& alsq2, const real& alsq2n,
         const real& aewald, const real& damp, const real& pgamma, const real& dscale, const real& pscale,
        real3& fid,real3& fip,real3& fkd,real3& fkp,real& d1, real& sc3, real& sc5, real& bn1, real& bn2, const int do_correct) {

   real exp2a;
   real invd1,invd2,invd3,invd5,invd7;
   real sc7;
   real drr3,drr5,drr7,prr3,prr5,prr7;
   real dir,qirr,dkr,qkrr;
   real qirx,qiry,qirz,qkrx,qkry,qkrz;
   real fkmx,fkmy,fkmz,fimx,fimy,fimz;
   real invdamp,expdamp1,damp1;
   real ralpha,bn0,bn3;
   const real one=1.;
   const real two=2.;

   damp1   = -100.0;
   invdamp = 1/damp;
   invd2   = 1/d2;
   d1      = f_sqrt(d2);
   invd1   = 1/d1;

   sc3     = one;
   sc5     = one;
   sc7     = one;

   invd3   = invd1  * invd2;
   invd5   = invd3  * invd2;
   invd7   = invd5  * invd2;

   if (damp!=0.0) damp1 = - pgamma*(d1*invdamp)*(d1*invdamp)*(d1*invdamp);

   if (damp1 > -50.0)  {
      expdamp1  = f_exp(damp1);
      sc3  = one - expdamp1;
      sc5  = one - expdamp1*(one - damp1);
      sc7  = one - expdamp1*(one - damp1 + 0.6*damp1*damp1);
   }

   if (do_correct) {
      /*  [dp]scale equal to 1-[dp]scale in this case */
      drr3    =      sc3*dscale * invd3;
      drr5    =  3 * sc5*dscale * invd5;
      drr7    = 15 * sc7*dscale * invd7;

      prr3    =      sc3*pscale * invd3;
      prr5    =  3 * sc5*pscale * invd5;
      prr7    = 15 * sc7*pscale * invd7;
   }
   else {

      /* calculate the error function damping terms */
      ralpha  = aewald * d1;
      exp2a   = f_exp( -ralpha*ralpha );
      bn0     = f_erfc(ralpha);  // becomes a macro with USE_ERFC_HASTINGS
 
      bn0     =    bn0                                     * invd1;
      bn1     = (  bn0  + alsq2             *alsq2n*exp2a) * invd2;
      bn2     = (3*bn1  + alsq2*alsq2       *alsq2n*exp2a) * invd2;
      bn3     = (5*bn2  + alsq2*alsq2*alsq2 *alsq2n*exp2a) * invd2;

      drr3    =      (one - sc3*dscale) * invd3;
      drr5    =  3 * (one - sc5*dscale) * invd5;
      drr7    = 15 * (one - sc7*dscale) * invd7;
      
      prr3    =      (one - sc3*pscale) * invd3;
      prr5    =  3 * (one - sc5*pscale) * invd5;
      prr7    = 15 * (one - sc7*pscale) * invd7;
   }
 
   /* compute some intermediate quantities */
   dir     = ip.dx *pos.x + ip.dy *pos.y + ip.dz *pos.z;
   qirx    = ip.qxx*pos.x + ip.qxy*pos.y + ip.qxz*pos.z;
   qiry    = ip.qxy*pos.x + ip.qyy*pos.y + ip.qyz*pos.z;
   qirz    = ip.qxz*pos.x + ip.qyz*pos.y + ip.qzz*pos.z;
   qirr    =   qirx*pos.x +   qiry*pos.y +   qirz*pos.z;

   dkr     = kp.dx *pos.x + kp.dy *pos.y +  kp.dz *pos.z;
   qkrx    = kp.qxx*pos.x + kp.qxy*pos.y +  kp.qxz*pos.z;
   qkry    = kp.qxy*pos.x + kp.qyy*pos.y +  kp.qyz*pos.z;
   qkrz    = kp.qxz*pos.x + kp.qyz*pos.y +  kp.qzz*pos.z;
   qkrr    =   qkrx*pos.x +   qkry*pos.y +    qkrz*pos.z;

   if (do_correct) {
      fimx = 0.0; fimy = 0.0; fimz = 0.0;
      fkmx = 0.0; fkmy = 0.0; fkmz = 0.0;
   }
   else {
      fimx = -( bn1*kp.c  - bn2*dkr + bn3*qkrr )*pos.x - bn1*kp.dx + two*bn2*qkrx;
      fimy = -( bn1*kp.c  - bn2*dkr + bn3*qkrr )*pos.y - bn1*kp.dy + two*bn2*qkry;
      fimz = -( bn1*kp.c  - bn2*dkr + bn3*qkrr )*pos.z - bn1*kp.dz + two*bn2*qkrz;
      fkmx =  ( bn1*ip.c  + bn2*dir + bn3*qirr )*pos.x - bn1*ip.dx - two*bn2*qirx;
      fkmy =  ( bn1*ip.c  + bn2*dir + bn3*qirr )*pos.y - bn1*ip.dy - two*bn2*qiry;
      fkmz =  ( bn1*ip.c  + bn2*dir + bn3*qirr )*pos.z - bn1*ip.dz - two*bn2*qirz;
   }

   fid.x  += fimx + ( drr3*kp.c  - drr5*dkr + drr7*qkrr )*pos.x +  drr3*kp.dx - two*drr5*qkrx;
   fid.y  += fimy + ( drr3*kp.c  - drr5*dkr + drr7*qkrr )*pos.y +  drr3*kp.dy - two*drr5*qkry;
   fid.z  += fimz + ( drr3*kp.c  - drr5*dkr + drr7*qkrr )*pos.z +  drr3*kp.dz - two*drr5*qkrz;
   fip.x  += fimx + ( prr3*kp.c  - prr5*dkr + prr7*qkrr )*pos.x +  prr3*kp.dx - two*prr5*qkrx;
   fip.y  += fimy + ( prr3*kp.c  - prr5*dkr + prr7*qkrr )*pos.y +  prr3*kp.dy - two*prr5*qkry;
   fip.z  += fimz + ( prr3*kp.c  - prr5*dkr + prr7*qkrr )*pos.z +  prr3*kp.dz - two*prr5*qkrz;

   fkd.x  += fkmx - ( drr3*ip.c  + drr5*dir + drr7*qirr )*pos.x +  drr3*ip.dx + two*drr5*qirx;
   fkd.y  += fkmy - ( drr3*ip.c  + drr5*dir + drr7*qirr )*pos.y +  drr3*ip.dy + two*drr5*qiry;
   fkd.z  += fkmz - ( drr3*ip.c  + drr5*dir + drr7*qirr )*pos.z +  drr3*ip.dz + two*drr5*qirz;
   fkp.x  += fkmx - ( prr3*ip.c  + prr5*dir + prr7*qirr )*pos.x +  prr3*ip.dx + two*prr5*qirx;
   fkp.y  += fkmy - ( prr3*ip.c  + prr5*dir + prr7*qirr )*pos.y +  prr3*ip.dy + two*prr5*qiry;
   fkp.z  += fkmz - ( prr3*ip.c  + prr5*dir + prr7*qirr )*pos.z +  prr3*ip.dz + two*prr5*qirz;

}

__global__ void check_loc( const int*restrict pglob, const int*restrict ipole, const int*restrict ploc, const int npolelocnlb, const int nbloc, const int rank ){
   for ( int ii=threadIdx.x + blockIdx.x*blockDim.x;ii<npolelocnlb;ii+=blockDim.x*gridDim.x ){
       const int iipole = pglob[ii];
       const int iploc  = ploc [ii];
       if (iploc==0 || iploc>nbloc ) printf("check_ploc(%d) pole(%d) glob(%d) Idx(%d) rank(%d)\n",iploc,ipole[ii],iipole,ii+1,rank);
       //iploc = ploc_s(iipole)
       //if (iploc==0 || iploc.gt.nbloc) printf("out ploc %d %d %d %d",iploc,iipole,i,rank);
   }
}

__global__
void cu_efld0_direct_core (EFLD0_PARAMS1){

   const int ithread = threadIdx.x + blockIdx.x*blockDim.x;
   const int iwarp   =               ithread / WARP_SIZE;
   const int nwarp   =  blockDim.x*gridDim.x / WARP_SIZE;
   const int ilane   = threadIdx.x & (WARP_SIZE-1);
   int accept_mid    = 1;

   int klane,srclane;
   int ii,j;
   int iblock,idx,kdx;
   int iipole,iglob,iploc,kpole,kglob,kploc,kglob_;
   int do_pair,same_block;
   real xk_,yk_,zk_,d2;
   real ipdp,ipgm,kpdp,kpgm,pdp,pgm;
   rpole_elt ip;
   real3 posi,pos;
   real3 fid,fip;
   __shared__ real3 posk[BLOCK_DIM],fkd[BLOCK_DIM],fkp[BLOCK_DIM];
   __shared__ rpole_elt kp[BLOCK_DIM];
   //__shared__ int ncalc[4];
   //__shared__ int cont;

   //if (ithread==0) printf( " %i %i %i %i %i %i %i " r_Format r_Format r_Format "\n", nwarp,nproc,npolelocnlb,npolebloc,n,npolelocnlb_pair,cut2,alsq2,aewald);

   for ( ii=iwarp; ii<npolelocnlb_pair; ii+=nwarp ){
      /*  Load atom block i parameters */
      iblock  = ieblst[ii];
      if (iblock==0) continue;
      idx     = (iblock-1)*WARP_SIZE + ilane;
      iipole  = pglob[idx] -1;
      iglob   = ipole[idx] -1;
      iploc   = ploc [idx] -1;
      posi.x  = x[idx];
      posi.y  = y[idx];
      posi.z  = z[idx];
      ipdp    = pdamp[iipole];
      ipgm    = thole[iipole];
      ip.c    = rpole[iipole][0];
      ip.dx   = rpole[iipole][1];
      ip.dy   = rpole[iipole][2];
      ip.dz   = rpole[iipole][3];
      ip.qxx  = rpole[iipole][4];
      ip.qxy  = rpole[iipole][5];
      ip.qxz  = rpole[iipole][6];
      ip.qyy  = rpole[iipole][8];
      ip.qyz  = rpole[iipole][9];
      ip.qzz  = rpole[iipole][12];

      /*  Load atom block k parameters */
      kdx     = eblst[ii*WARP_SIZE+ ilane] -1;
      kpole   = pglob[kdx] -1;
      kglob   = ipole[kdx] -1;
      kploc   = ploc [kdx] -1;
      posk[threadIdx.x].x  = x[kdx];
      posk[threadIdx.x].y  = y[kdx];
      posk[threadIdx.x].z  = z[kdx];
      kpdp    = pdamp[kpole];
      kpgm    = thole[kpole];
      kp[threadIdx.x].c    = rpole[kpole][0];
      kp[threadIdx.x].dx   = rpole[kpole][1];
      kp[threadIdx.x].dy   = rpole[kpole][2];
      kp[threadIdx.x].dz   = rpole[kpole][3];
      kp[threadIdx.x].qxx  = rpole[kpole][4];
      kp[threadIdx.x].qxy  = rpole[kpole][5];
      kp[threadIdx.x].qxz  = rpole[kpole][6];
      kp[threadIdx.x].qyy  = rpole[kpole][8];
      kp[threadIdx.x].qyz  = rpole[kpole][9];
      kp[threadIdx.x].qzz  = rpole[kpole][12];
      //if (ilane==1) ncalc[threadIdx.x/WARP_SIZE]=0;

      /* set compute Data to]0 */
      fid.x   = 0;
      fid.y   = 0;
      fid.z   = 0;
      fip.x   = 0;
      fip.y   = 0;
      fip.z   = 0;
      fkd[threadIdx.x].x = 0;
      fkd[threadIdx.x].y = 0;
      fkd[threadIdx.x].z = 0;
      fkp[threadIdx.x].x = 0;
      fkp[threadIdx.x].y = 0;
      fkp[threadIdx.x].z = 0;
      //cont=0;

      same_block = ( idx!=kdx )? 0:1 ;

      for ( j=0; j<WARP_SIZE; j++ ){
         srclane  = (ilane+j) & (WARP_SIZE-1);
         klane    = threadIdx.x-ilane + srclane;
         kglob_   =      __shfl_sync(ALL_LANES,kglob ,srclane);
         pdp      = ipdp*__shfl_sync(ALL_LANES,kpdp  ,srclane);
         pgm      =      __shfl_sync(ALL_LANES,kpgm  ,srclane);
         if (ipgm<pgm) pgm = ipgm;

         if (nproc>1 && balanced ) {
            xk_   = posk[klane].x;
            yk_   = posk[klane].y;
            zk_   = posk[klane].z;
            pos.x = posi.x - xk_;
            pos.y = posi.y - yk_;
            pos.z = posi.z - zk_;
            accept_mid = Midpointimage(xk_,yk_,zk_,pos.x,pos.y,pos.z);
            pos.x=-pos.x; pos.y=-pos.y; pos.z=-pos.z;
         }
         else {
            pos.x = posk[klane].x - posi.x;
            pos.y = posk[klane].y - posi.y;
            pos.z = posk[klane].z - posi.z;
            Image(pos.x,pos.y,pos.z);
         }
         d2      = pos.x*pos.x + pos.y*pos.y + pos.z*pos.z;
         do_pair = (same_block)? (iglob < kglob_):1;

         if (do_pair && d2<=cut2 && accept_mid) {
            //atomicAdd( &cont, 1);
            /* Compute one interaction
               Since the interaction is not symetrical we need to switch comput when necessary */
            real d,sc3,sc5,bn1,bn2;
            //if (iblock<500) atomicAdd( &ncalc[threadIdx.x/WARP_SIZE],1 );
            if (iglob<kglob_)
               efld0_couple(d2,pos,ip,kp[klane],alsq2,alsq2n,aewald,pdp,pgm,1.0,1.0,
                            fid,fip,fkd[klane],fkp[klane],d,sc3,sc5,bn1,bn2,0);
            else{
               pos.x = -pos.x; pos.y=-pos.y; pos.z=-pos.z;
               efld0_couple(d2,pos,kp[klane],ip,alsq2,alsq2n,aewald,pdp,pgm,1.0,1.0,
                            fkd[klane],fkp[klane],fid,fip,d,sc3,sc5,bn1,bn2,0);
            }
         }
      }

      //if (ilane==0) printf("%d  %d  %d \n",ii,iblock,cont);
      /* increment electric field for each atoms */
      atomicAdd( &efi[iploc][0],fid.x );
      atomicAdd( &efi[iploc][1],fid.y );
      atomicAdd( &efi[iploc][2],fid.z );
      atomicAdd( &efi[iploc][3],fip.x );
      atomicAdd( &efi[iploc][4],fip.y );
      atomicAdd( &efi[iploc][5],fip.z );
      atomicAdd( &efi[kploc][0],fkd[threadIdx.x].x );
      atomicAdd( &efi[kploc][1],fkd[threadIdx.x].y );
      atomicAdd( &efi[kploc][2],fkd[threadIdx.x].z );
      atomicAdd( &efi[kploc][3],fkp[threadIdx.x].x );
      atomicAdd( &efi[kploc][4],fkp[threadIdx.x].y );
      atomicAdd( &efi[kploc][5],fkp[threadIdx.x].z );
      __syncwarp(ALL_LANES);
      //if (ilane==1 && iblock<500) printf("ii %8d %8d    %8d \n",ii+1,iblock,ncalc[threadIdx.x/WARP_SIZE]);
   }
}

__global__
void cu_otfdc_efld0_direct_core (OTFDC_EFLD0_PARAMS1){

   const int ithread = threadIdx.x + blockIdx.x*blockDim.x;
   const int iwarp   =               ithread / WARP_SIZE;
   const int nwarp   =  blockDim.x*gridDim.x / WARP_SIZE;
   const int ilane   = threadIdx.x & (WARP_SIZE-1);
   int accept_mid    = 1;

   int klane,srclane;
   int ii,j;
   int iblock,idx,kdx,li,atii,maxrow,kofi;
   int iipole,iglob,iploc,kpole,kglob,kploc,kglob_;
   int do_pair,same_block;
   real xk_,yk_,zk_,d2;
   real ipdp,ipgm,kpdp,kpgm,pdp,pgm;
   rpole_elt ip;
   real3 posi,pos;
   real3 fid,fip;
   __shared__ int lk[BLOCK_DIM];
   __shared__ real3 posk[BLOCK_DIM],fkd[BLOCK_DIM],fkp[BLOCK_DIM];
   __shared__ rpole_elt kp[BLOCK_DIM];

   //if (ithread==0) printf( " %i %i %i %i %i %i %i " r_Format r_Format r_Format "\n", nwarp,nproc,npolelocnlb,npolebloc,n,npolelocnlb_pair,cut2,alsq2,aewald);

   for ( ii=iwarp; ii<npolelocnlb_pair; ii+=nwarp ){
      /*  Load atom block i parameters */
      iblock  = ieblst[ii];
      if (iblock==0) continue;
      idx     = (iblock-1)*WARP_SIZE + ilane;
      iipole  = pglob[idx] -1;
      iglob   = ipole[idx] -1;
      iploc   = ploc [idx] -1;
      posi.x  = x[idx];
      posi.y  = y[idx];
      posi.z  = z[idx];
      // gather zmat data 
      li      =   grplst[iglob];
      atii    = (atmofst[iglob]-1)*3;
      maxrow  =  npergrp[li-1]*3;
      kofi    =    kofst[li-1];

      ipdp    = pdamp[iipole];
      ipgm    = thole[iipole];
      ip.c    = rpole[iipole][0];
      ip.dx   = rpole[iipole][1];
      ip.dy   = rpole[iipole][2];
      ip.dz   = rpole[iipole][3];
      ip.qxx  = rpole[iipole][4];
      ip.qxy  = rpole[iipole][5];
      ip.qxz  = rpole[iipole][6];
      ip.qyy  = rpole[iipole][8];
      ip.qyz  = rpole[iipole][9];
      ip.qzz  = rpole[iipole][12];

      /*  Load atom block k parameters */
      kdx     = eblst[ii*WARP_SIZE+ ilane] -1;
      kpole   = pglob[kdx] -1;
      kglob   = ipole[kdx] -1;
      kploc   = ploc [kdx] -1;
      lk  [threadIdx.x]    = grplst[kglob];
      posk[threadIdx.x].x  = x[kdx];
      posk[threadIdx.x].y  = y[kdx];
      posk[threadIdx.x].z  = z[kdx];
      kpdp    = pdamp[kpole];
      kpgm    = thole[kpole];
      kp[threadIdx.x].c    = rpole[kpole][0];
      kp[threadIdx.x].dx   = rpole[kpole][1];
      kp[threadIdx.x].dy   = rpole[kpole][2];
      kp[threadIdx.x].dz   = rpole[kpole][3];
      kp[threadIdx.x].qxx  = rpole[kpole][4];
      kp[threadIdx.x].qxy  = rpole[kpole][5];
      kp[threadIdx.x].qxz  = rpole[kpole][6];
      kp[threadIdx.x].qyy  = rpole[kpole][8];
      kp[threadIdx.x].qyz  = rpole[kpole][9];
      kp[threadIdx.x].qzz  = rpole[kpole][12];

      /* set compute Data to]0 */
      fid.x   = 0;
      fid.y   = 0;
      fid.z   = 0;
      fip.x   = 0;
      fip.y   = 0;
      fip.z   = 0;
      fkd[threadIdx.x].x = 0;
      fkd[threadIdx.x].y = 0;
      fkd[threadIdx.x].z = 0;
      fkp[threadIdx.x].x = 0;
      fkp[threadIdx.x].y = 0;
      fkp[threadIdx.x].z = 0;

      same_block = ( idx!=kdx )? 0:1 ;

      for ( j=0; j<WARP_SIZE; j++ ){
         srclane  = (ilane+j) & (WARP_SIZE-1);
         klane    = threadIdx.x-ilane + srclane;
         kglob_   =      __shfl_sync(ALL_LANES,kglob ,srclane);
         pdp      = ipdp*__shfl_sync(ALL_LANES,kpdp  ,srclane);
         pgm      =      __shfl_sync(ALL_LANES,kpgm  ,srclane);
         if (ipgm<pgm) pgm = ipgm;

         if (nproc>1) {
            xk_   = posk[klane].x;
            yk_   = posk[klane].y;
            zk_   = posk[klane].z;
            pos.x = posi.x - xk_;
            pos.y = posi.y - yk_;
            pos.z = posi.z - zk_;
            accept_mid = Midpointimage(xk_,yk_,zk_,pos.x,pos.y,pos.z);
            pos.x=-pos.x; pos.y=-pos.y; pos.z=-pos.z;
         }
         else {
            pos.x = posk[klane].x - posi.x;
            pos.y = posk[klane].y - posi.y;
            pos.z = posk[klane].z - posi.z;
            Image(pos.x,pos.y,pos.z);
         }
         d2      = pos.x*pos.x + pos.y*pos.y + pos.z*pos.z;
         do_pair = (same_block)? (iglob < kglob_):1;

         if (do_pair && d2<=cut2 && accept_mid) {
            /* Compute one interaction
               Since the interaction is not symetrical we need to switch comput when necessary */
            real d,sc3,sc5,bn1,bn2;
            if (iglob<kglob_)
               efld0_couple(d2,pos,ip,kp[klane],alsq2,alsq2n,aewald,pdp,pgm,1.0,1.0,
                            fid,fip,fkd[klane],fkp[klane],d,sc3,sc5,bn1,bn2,0);
            else{
               pos.x = -pos.x; pos.y=-pos.y; pos.z=-pos.z;
               efld0_couple(d2,pos,kp[klane],ip,alsq2,alsq2n,aewald,pdp,pgm,1.0,1.0,
                            fkd[klane],fkp[klane],fid,fip,d,sc3,sc5,bn1,bn2,0);
            }
            if (li==lk[klane] && li!=-1) {

               bn1    = bn1 -     (1.0 - sc3)/ (d*d2);
               bn2    = bn2 - 3.0*(1.0 - sc5)/ (d*d2*d2);
               int atkk = (atmofst[kglob_] - 1)*3;
               int cofst1,cofst2,cofst3,rofst;
               if (atii < atkk) {
                 cofst1 = atii + 1;
                 cofst2 = atii + 2;
                 cofst3 = atii + 3;
                 rofst  = atkk;
               } else {
                 cofst1 = atkk + 1;
                 cofst2 = atkk + 2;
                 cofst3 = atkk + 3;
                 rofst  = atii;
               }

               cofst1 = (cofst1-1)*(2*maxrow-cofst1)/2;
               cofst2 = (cofst2-1)*(2*maxrow-cofst2)/2;
               cofst3 = (cofst3-1)*(2*maxrow-cofst3)/2;
               zmat[rofst+0+cofst1+kofi] =  bn1 - bn2*pos.x*pos.x;
               zmat[rofst+1+cofst1+kofi] =      - bn2*pos.x*pos.y;
               zmat[rofst+2+cofst1+kofi] =      - bn2*pos.x*pos.z;
               zmat[rofst+0+cofst2+kofi] =      - bn2*pos.x*pos.y;
               zmat[rofst+1+cofst2+kofi] =  bn1 - bn2*pos.y*pos.y;
               zmat[rofst+2+cofst2+kofi] =      - bn2*pos.y*pos.z;
               zmat[rofst+0+cofst3+kofi] =      - bn2*pos.x*pos.z;
               zmat[rofst+1+cofst3+kofi] =      - bn2*pos.y*pos.z;
               zmat[rofst+2+cofst3+kofi] =  bn1 - bn2*pos.z*pos.z;

            }
         }
      }

      /* increment electric field for each atoms */
      atomicAdd( &efi[iploc][0],fid.x );
      atomicAdd( &efi[iploc][1],fid.y );
      atomicAdd( &efi[iploc][2],fid.z );
      atomicAdd( &efi[iploc][3],fip.x );
      atomicAdd( &efi[iploc][4],fip.y );
      atomicAdd( &efi[iploc][5],fip.z );
      atomicAdd( &efi[kploc][0],fkd[threadIdx.x].x );
      atomicAdd( &efi[kploc][1],fkd[threadIdx.x].y );
      atomicAdd( &efi[kploc][2],fkd[threadIdx.x].z );
      atomicAdd( &efi[kploc][3],fkp[threadIdx.x].x );
      atomicAdd( &efi[kploc][4],fkp[threadIdx.x].y );
      atomicAdd( &efi[kploc][5],fkp[threadIdx.x].z );
   }
}


__global__
void cu_tmatxb_pme_core (TMATXB_PARAMS1){

   const int ithread = threadIdx.x + blockIdx.x*blockDim.x;
   const int iwarp   =               ithread / WARP_SIZE;
   const int nwarp   =  blockDim.x*gridDim.x / WARP_SIZE;
   const int ilane   = threadIdx.x & (WARP_SIZE-1);
   int accept_mid    = 1;

   int klane,srclane;
   int ii,j;
   int iblock,idx,kdx;
   int iipole,iglob,iploc,kpole,kploc;
   int do_pair,same_block;
   real xk_,yk_,zk_,d2;
   real ipdp,ipgm,pdp,pgm;
   real6 dpuk_;
   real3 pos;
   __shared__ int kglob[BLOCK_DIM], ikstat[BLOCK_DIM];
   __shared__ real kpdp[BLOCK_DIM],kpgm[BLOCK_DIM];
   __shared__ real3 posk[BLOCK_DIM],posi[BLOCK_DIM];
   __shared__ real3 fkd[BLOCK_DIM],fkp[BLOCK_DIM];
   __shared__ real3 fid[BLOCK_DIM],fip[BLOCK_DIM];
   __shared__ real6 dpui[BLOCK_DIM],dpuk[BLOCK_DIM];


   //if (ithread==0) printf( " %i %i %i %i %i %i %i " r_Format r_Format r_Format "\n", nwarp,nproc,npolelocnlb,npolebloc,n,npolelocnlb_pair,cut2,alsq2,aewald);

   for ( ii=iwarp; ii<npolelocnlb_pair; ii+=nwarp ){
      /*  Load atom block i parameters */
      iblock  = ieblst[ii];
      if (iblock==0) continue;
      idx     = (iblock-1)*WARP_SIZE + ilane;
      iipole  = pglob[idx] -1;
      iglob   = ipole[idx] -1;
      iploc   = ploc [idx] -1;
      posi[threadIdx.x].x  = x[idx];
      posi[threadIdx.x].y  = y[idx];
      posi[threadIdx.x].z  = z[idx];
      ipdp    = pdamp[iipole];
      ipgm    = thole[iipole];
      dpui[threadIdx.x].x  = mu[iploc][0];
      dpui[threadIdx.x].y  = mu[iploc][1];
      dpui[threadIdx.x].z  = mu[iploc][2];
      dpui[threadIdx.x].xx = mu[iploc][3];
      dpui[threadIdx.x].yy = mu[iploc][4];
      dpui[threadIdx.x].zz = mu[iploc][5];

      /*  Load atom block k parameters */
      kdx     = eblst[ii*WARP_SIZE+ ilane] -1;
      kpole   = pglob[kdx] -1;
      kglob[threadIdx.x]   = ipole[kdx] -1;
      kploc   = ploc [kdx] -1;
      posk[threadIdx.x].x  = x[kdx];
      posk[threadIdx.x].y  = y[kdx];
      posk[threadIdx.x].z  = z[kdx];
      kpdp[threadIdx.x]    = pdamp[kpole];
      kpgm[threadIdx.x]    = thole[kpole];
      dpuk[threadIdx.x].x  = mu[kploc][0];
      dpuk[threadIdx.x].y  = mu[kploc][1];
      dpuk[threadIdx.x].z  = mu[kploc][2];
      dpuk[threadIdx.x].xx = mu[kploc][3];
      dpuk[threadIdx.x].yy = mu[kploc][4];
      dpuk[threadIdx.x].zz = mu[kploc][5];

      /* set compute Data to]0 */
      ikstat[threadIdx.x]   = 0;
      fid   [threadIdx.x].x = 0;
      fid   [threadIdx.x].y = 0;
      fid   [threadIdx.x].z = 0;
      fip   [threadIdx.x].x = 0;
      fip   [threadIdx.x].y = 0;
      fip   [threadIdx.x].z = 0;
      fkd   [threadIdx.x].x = 0;
      fkd   [threadIdx.x].y = 0;
      fkd   [threadIdx.x].z = 0;
      fkp   [threadIdx.x].x = 0;
      fkp   [threadIdx.x].y = 0;
      fkp   [threadIdx.x].z = 0;

      same_block = ( idx!=kdx )? 0:1 ;

      #pragma unroll
      for ( int i=0; i<2; i++ ){
         srclane     = (ilane+i) & (WARP_SIZE-1);
         int iilane  = threadIdx.x-ilane + srclane;
      for ( j=0; j<WARP_SIZE; j++ ){
         if (atomicOr( &ikstat[iilane],1<<j ) & 1<<j) continue;

         srclane  = (ilane+j) & (WARP_SIZE-1);
         klane    = threadIdx.x-ilane + srclane;
         dpuk_.x  = dpuk[klane].x ;
         dpuk_.y  = dpuk[klane].y ;
         dpuk_.z  = dpuk[klane].z ;
         dpuk_.xx = dpuk[klane].xx;
         dpuk_.yy = dpuk[klane].yy;
         dpuk_.zz = dpuk[klane].zz;
         pdp      = ipdp*kpdp[klane];
         pgm      =      kpgm[klane];
         if (ipgm<pgm) pgm = ipgm;

         if (nproc>1) {
            xk_   = posk[klane].x;
            yk_   = posk[klane].y;
            zk_   = posk[klane].z;
            pos.x = posi[iilane].x - xk_;
            pos.y = posi[iilane].y - yk_;
            pos.z = posi[iilane].z - zk_;
            accept_mid = Midpointimage(xk_,yk_,zk_,pos.x,pos.y,pos.z);
         }
         else {
            pos.x = posi[iilane].x - posk[klane].x;
            pos.y = posi[iilane].y - posk[klane].y;
            pos.z = posi[iilane].z - posk[klane].z;
            Image(pos.x,pos.y,pos.z);
         }
         d2      = pos.x*pos.x + pos.y*pos.y + pos.z*pos.z;
         do_pair = (same_block)? iglob < kglob[klane] : 1 ;

         if (do_pair && d2<=cut2 && accept_mid) {
             /* compute one interaction */
             tmatxb_couple(d2,pos,dpui[iilane],dpuk_,pdp,pgm,aewald,alsq2,alsq2n,1.
                          ,fid[iilane],fip[iilane],fkd[klane],fkp[klane]);
         }
      }
      }
      __syncwarp(ALL_LANES);
      /*if (ii==0&&ilane==3) printf (" %5d %5d %5i %7d " r10_Format "\n" ,
         ii,iglob,klane,kdx,dpui.x,dpuk_.x,dpui.y,dpuk_.y,
         fid.x,fid.y,fip.x); */

      /* increment electric field for each atoms */
      atomicAdd( &efi[iploc][0],fid[threadIdx.x].x );
      atomicAdd( &efi[iploc][1],fid[threadIdx.x].y );
      atomicAdd( &efi[iploc][2],fid[threadIdx.x].z );
      atomicAdd( &efi[iploc][3],fip[threadIdx.x].x );
      atomicAdd( &efi[iploc][4],fip[threadIdx.x].y );
      atomicAdd( &efi[iploc][5],fip[threadIdx.x].z );
      atomicAdd( &efi[kploc][0],fkd[threadIdx.x].x );
      atomicAdd( &efi[kploc][1],fkd[threadIdx.x].y );
      atomicAdd( &efi[kploc][2],fkd[threadIdx.x].z );
      atomicAdd( &efi[kploc][3],fkp[threadIdx.x].x );
      atomicAdd( &efi[kploc][4],fkp[threadIdx.x].y );
      atomicAdd( &efi[kploc][5],fkp[threadIdx.x].z );
   }
}


EXTERN_C_BEG

int nproc      = 1;
int rank       = 0;
int devicenum  = 0;
int tinkerdebug= 0;
hipDeviceProp_t devProp;

void C_init_env (int devicenum_=0, int nproc_=1, int rank_=0, int tinkerdebug_=0){
   hipGetDeviceProperties(&devProp,devicenum_);
   devicenum   = devicenum_;
   nproc       = nproc_;
   rank        = rank_;
   tinkerdebug = tinkerdebug_;
}

real _xcell;
real _ycell;
real _zcell;
real _ixcell;
real _iycell;
real _izcell;
real eps_cell;
real _box34;
int octahedron;

void C_get_cell( real xcell_, real ycell_, real zcell_, real eps_cell_, int octa_, real box34_ ){
   _xcell   = xcell_;
   _ycell   = ycell_;
   _zcell   = zcell_;
   _ixcell  = (real) 1.0/ (double)xcell_;
   _iycell  = (real) 1.0/ (double)ycell_;
   _izcell  = (real) 1.0/ (double)zcell_;
   eps_cell = eps_cell_;
   octahedron = octa_;
   _box34   = box34_;
}

int dynamic_gS        = 1;
int first_call_efld0  = 1;
int gS_efld    = 160;
int gS_loc     = 160;
const int maxBlock = 1<<16;


void cu_efld0_direct(EFLD0_PARAMS,hipStream_t st){
   const int sh = 0;
   hipError_t ierrSync;

   if (first_call_efld0){
      first_call_efld0=0;
      cudaKernelMaxGridSize(gS_efld,cu_efld0_direct_core,BLOCK_DIM,0)  /* This a Macro Function */
      if(rank==0 && tinkerdebug&1) {
         printf (" gridSize efld0     %d \n", gS_efld);
         printf (" balanced computation     %d \n ", balanced);
      }
      if (nproc>1) {
         cudaKernelMaxGridSize(gS_loc,check_loc,BLOCK_DIM,0)  /* This a Macro Function */
         check_loc<<<gS_loc,BLOCK_DIM,sh,st>>>(pglob,ipole,ploc,npolelocnlb,npolebloc,rank);
         ierrSync = tinkerdebug ? hipDeviceSynchronize() : hipGetLastError();
         if (ierrSync != hipSuccess) printf("check_loc kernel error: %d ( %s )\n",ierrSync,hipGetErrorString(ierrSync));
      }
   }

   if (dynamic_gS) gS_efld= (npolelocnlb_pair>>2 < maxBlock) ? npolelocnlb_pair>>2 : maxBlock ;
   cu_efld0_direct_core<<<gS_efld,BLOCK_DIM,sh,st>>> (EFLD0_ARGS);
   ierrSync = tinkerdebug ? hipDeviceSynchronize() : hipGetLastError();
   if (ierrSync != hipSuccess) printf("efld0_direct_core C kernel error: %d ( %s )\n",ierrSync,hipGetErrorString(ierrSync));

   return;
}

int first_call_otfdc_efld0  = 1;
int gS_otfdc_efld           = 160;

void cu_otfdc_efld0_direct(OTFDC_EFLD0_PARAMS,hipStream_t st){
   const int sh = 0;

   if (first_call_otfdc_efld0){
      first_call_otfdc_efld0=0;
      cudaKernelMaxGridSize(gS_otfdc_efld,cu_otfdc_efld0_direct_core,BLOCK_DIM,0)  /* This a Macro Function */
      if (rank==0 && tinkerdebug&1) {
         printf (" gridSize oftdc_efld0     %d \n", gS_efld);
      }
   }
   if (dynamic_gS) gS_otfdc_efld= npolelocnlb_pair/4;

   cu_otfdc_efld0_direct_core<<<gS_otfdc_efld,BLOCK_DIM,sh,st>>> (OTFDC_EFLD0_ARGS);
   if  (tinkerdebug) gpuErrchk( hipDeviceSynchronize() )
   else              gpuErrchk( hipGetLastError() )
   return;
}

int first_call_tmatxb = 1;
int gS_tmat           = 160;

void cu_tmatxb_pme(TMATXB_PARAMS,hipStream_t st){
   //int gS = 160;
   const int sh = 0;

   if (first_call_tmatxb){
      first_call_tmatxb = 0;
      cudaKernelMaxGridSize(gS_tmat,cu_tmatxb_pme_core,BLOCK_DIM,0)  /* This a Macro Function */
      if (rank==0 && tinkerdebug&1) printf (" gridSize tmatxb_cu %d \n", gS_tmat);
   }
   if (dynamic_gS) gS_tmat= npolelocnlb_pair/8;

   cu_tmatxb_pme_core<<<gS_tmat,BLOCK_DIM,sh,st>>> (TMATXB_ARGS);
   hipError_t ierrSync;
   if(tinkerdebug) ierrSync = hipDeviceSynchronize();
   else            ierrSync = hipGetLastError();
   if (ierrSync != hipSuccess)
      printf("tmatxb_pme_core C kernel error: %d \n  %s",ierrSync, hipGetErrorString(ierrSync));
   return;
}
EXTERN_C_END
