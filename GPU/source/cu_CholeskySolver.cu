#include "hip/hip_runtime.h"
#include "utils.h"

#ifdef SINGLE
#   define cuPOTRF_buffSize hipsolverDnSpotrf_bufferSize
#   if (CUDART_VERSION>10010)
#     define cuGESV_buffSize  hipsolverDnSSgesv_bufferSize
#     define cuGESV           hipsolverDnSSgesv
#     define cuGESVm_buffSize hipsolverDnSSgesv_bufferSize
#     define cuGESVm          hipsolverDnSSgesv
#   else
#     define cuGETRF_buffsize  hipsolverDnSgetrf_bufferSize
#     define cuGETRF           hipsolverDnSgetrf
#     define cuGETRS           hipsolverDnSgetrs
#     define cuGETRFm_buffsize hipsolverDnSgetrf_bufferSize
#     define cuGETRFm          hipsolverDnSgetrf
#     define cuGETRSm          hipsolverDnSgetrs
#   endif
#   define cuPOTRF_buffSize hipsolverDnSpotrf_bufferSize
#   define cuPOTRF          hipsolverDnSpotrf
#   define cuPOTRS          hipsolverDnSpotrs
#elif defined(MIXED)
#   if (CUDART_VERSION>10010)
#     define cuGESV_buffSize  hipsolverDnSSgesv_bufferSize
#     define cuGESV           hipsolverDnSSgesv
#     define cuGESVm_buffSize hipsolverDnDDgesv_bufferSize
#     define cuGESVm          hipsolverDnDDgesv
#   else
#     define cuGETRF_buffsize  hipsolverDnSgetrf_bufferSize
#     define cuGETRF           hipsolverDnSgetrf
#     define cuGETRS           hipsolverDnSgetrs
#     define cuGETRFm_buffsize hipsolverDnDgetrf_bufferSize
#     define cuGETRFm          hipsolverDnDgetrf
#     define cuGETRSm          hipsolverDnDgetrs
#   endif
#   define cuPOTRF_buffSize    hipsolverDnSpotrf_bufferSize
#   define cuPOTRF             hipsolverDnSpotrf
#   define cuPOTRS             hipsolverDnSpotrs
#else
#   define cuPOTRF_buffSize hipsolverDnDpotrf_bufferSize
#   if (CUDART_VERSION>10010)
#     define cuGESV_buffSize  hipsolverDnDDgesv_bufferSize
#     define cuGESV           hipsolverDnDDgesv
#     define cuGESVm_buffSize hipsolverDnDDgesv_bufferSize
#     define cuGESVm          hipsolverDnDDgesv
#   else
#     define cuGETRF_buffsize  hipsolverDnDgetrf_bufferSize
#     define cuGETRF           hipsolverDnDgetrf
#     define cuGETRS           hipsolverDnDgetrs
#     define cuGETRFm_buffsize hipsolverDnDgetrf_bufferSize
#     define cuGETRFm          hipsolverDnDgetrf
#     define cuGETRSm          hipsolverDnDgetrs
#   endif
#   define cuPOTRF_buffSize    hipsolverDnDpotrf_bufferSize
#   define cuPOTRF             hipsolverDnDpotrf
#   define cuPOTRS             hipsolverDnDpotrs
#endif

extern const int rank;
extern const int tinkerdebug;

/* ---------
   Cu Solver global environnement
   ---------
*/
hipsolverHandle_t cuCholHandle = NULL;
const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
real* d_workSpace=NULL;
realm* dm_workSpace=NULL;
real* d_inB=NULL;
realm* dm_inB=NULL;
size_t s_workSpaceSize=0,sm_workSpaceSize=0;
size_t s_inB=0,sm_inB=0;
int info;
int* d_info;

__global__ void CheckcuSolverInfo (int* d_info, int line, int rank) {
  if (*d_info != 0) printf (" Error info %d with cuSolver in " __FILE__ " :%d \n", *d_info, line);
}

EXTERN_C_BEG

void initcuSolverHandle(hipStream_t stream){

   if (cuCholHandle) {
      printf("\n WARNING ! CuSolver Handle has already been initialized");
      return;
   }

   gpuErrchkSolver( hipsolverDnCreate(&cuCholHandle) )
   gpuErrchkSolver( hipsolverSetStream(cuCholHandle, stream) )

   gpuErrchk( hipMalloc (&d_info, sizeof(int)) )
   if (rank==0) printf ("\n *** Using CuSolver Library ***\n\n" );
}

/* ----------------
   Reallocation procedure based on MOD_utilgpu.f reallocate_acc
   ---------------- */
void device_reallocate(void** array, const size_t bytesSize, size_t& PrevSize){
   if ( !(*array) ){
      gpuErrchk( hipMalloc(array,bytesSize) )
      PrevSize = bytesSize;
      //printf(" device_reallocate size %lu \n", PrevSize);
   }
   else {
      if (bytesSize > PrevSize) {
         gpuErrchk( hipFree( *array ) )
         gpuErrchk( hipMalloc(array,bytesSize) )
         PrevSize = bytesSize;
         /*printf(" device_reallocate size %lu \n", PrevSize);*/
      }
   }
}

void p_pointer(void* ptr){
   printf(" %p \n", ptr);
}

void cuPOTRF_Wrapper(const int n, real* A, const int lda, hipStream_t stream){
   int Lwork=0;
   hipsolverStatus_t status1;

   gpuErrchk( cuPOTRF_buffSize(cuCholHandle, uplo, n, A, lda, &Lwork) )

   device_reallocate((void**)&d_workSpace, (size_t)Lwork*sizeof(real), s_workSpaceSize);

   status1 = cuPOTRF(cuCholHandle,uplo, n, A, lda, d_workSpace, Lwork, d_info);
   if (status1!=HIPSOLVER_STATUS_SUCCESS) printf( "Cholesky Factorisation on device failed with Error %d \n",status1 );

   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }

}

void cuPOTRS_Wrapper(const int n, real* A, const int lda, real* B, const int ldb, hipStream_t stream){
   hipsolverStatus_t status1;

   status1 = cuPOTRS(cuCholHandle, uplo, n, 1, A, lda, B, ldb, d_info);
   if (status1!=HIPSOLVER_STATUS_SUCCESS) printf( "Error %d solving Linear system \n",status1);

   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }
}

__global__ void printAB( real* A, real* B , int nrhs, int lwork_bytes, int n){
   printf(" cuGESV_Wrapper wsSize(%d) nrhs(%d) n(%d)\n",lwork_bytes,nrhs,n);
   printf(" Mat");
   for (int i=0; i<4; i++) printf(" %f ", A[i]);
   printf("\n");
   printf(" Vec");
   for (int i=0; i<2*nrhs; i++) printf(" %f ", B[i]);
   printf("\n");
}

__global__ void printS( real* A, real* B , int nrhs, int iter){
// printf(" Mat"); for (int i=0; i<4; i++) printf(" %f ", A[i]);
// printf("\n");
   printf(" Sol"); for (int i=0; i<2*nrhs; i++) printf(" %f ", B[i]);
   printf(" iter(%d)\n",iter);
}

void cuGESV_Wrapper(const int n, const int nrhs, real* A, const int lda, int* Ipiv, real* B, const int ldb, hipStream_t stream){
#if (CUDART_VERSION>10010)
   size_t lwork_bytes=0;
   int iter=0;
   size_t Bsize=nrhs*n*sizeof(real);
   gpuErrchkSolver( cuGESV_buffSize(cuCholHandle, n, nrhs, A, lda, Ipiv, d_inB, ldb, B, ldb, d_workSpace, &lwork_bytes) )
   device_reallocate((void**)&d_workSpace, (size_t)lwork_bytes, s_workSpaceSize);
   device_reallocate((void**)&d_inB, Bsize, s_inB);

   gpuErrchk( hipMemcpyAsync( d_inB,B,Bsize,hipMemcpyDeviceToDevice,stream ) )
   //printAB<<<1,1,0,stream>>>(A,B,nrhs,lwork_bytes,n);
   gpuErrchkSolver( cuGESV(cuCholHandle, n, nrhs, A, lda, Ipiv, d_inB, ldb, B, ldb, d_workSpace, lwork_bytes, &iter, d_info) )
   //printS <<<1,1,0,stream>>>(A,B,nrhs,iter);
#else
   int Lwork=0;
   gpuErrchkSolver( cuGETRF_buffsize(cuCholHandle, n, n, A, lda, &Lwork) )
   //printf(" LU solve n %d nrhs %d lda %d ldb %d Lwork %d\n",n,nrhs,lda,ldb, Lwork);
   device_reallocate((void**)&d_workSpace, (size_t)Lwork*sizeof(real), s_workSpaceSize);

   gpuErrchkSolver( cuGETRF(cuCholHandle, n, n, A, lda, d_workSpace, Ipiv, d_info) )
   gpuErrchkSolver( cuGETRS(cuCholHandle, HIPBLAS_OP_N, n, nrhs, A, lda, Ipiv, B, ldb, d_info) )
#endif
   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }
}

void cuGESVm_Wrapper(const int n, const int nrhs, realm* A, const int lda, int* Ipiv, realm* B, const int ldb, hipStream_t stream){
#if (CUDART_VERSION>10010)
   size_t lwork_bytes=0;
   int iter=0;
   size_t Bsize=nrhs*n*sizeof(realm);
   gpuErrchkSolver( cuGESVm_buffSize(cuCholHandle, n, nrhs, A, lda, Ipiv, dm_inB, ldb, B, ldb, dm_workSpace, &lwork_bytes) )
   device_reallocate((void**)&dm_workSpace, (size_t)lwork_bytes, sm_workSpaceSize);
   device_reallocate((void**)&dm_inB, Bsize, sm_inB);

   gpuErrchk( hipMemcpyAsync( dm_inB,B,Bsize,hipMemcpyDeviceToDevice,stream ) )
   //printAB<<<1,1,0,stream>>>(A,B,nrhs,lwork_bytes,n);
   gpuErrchkSolver( cuGESVm(cuCholHandle, n, nrhs, A, lda, Ipiv, dm_inB, ldb, B, ldb, dm_workSpace, lwork_bytes, &iter, d_info) )
   //printS <<<1,1,0,stream>>>(A,B,nrhs,iter);
#else
   int Lwork=0;
   gpuErrchkSolver( cuGETRFm_buffsize(cuCholHandle, n, n, A, lda, &Lwork) )
   //printf(" LU solve n %d nrhs %d lda %d ldb %d Lwork %d\n",n,nrhs,lda,ldb, Lwork);
   device_reallocate((void**)&dm_workSpace, (size_t)Lwork*sizeof(realm), sm_workSpaceSize);

   gpuErrchkSolver( cuGETRFm(cuCholHandle, n, n, A, lda, dm_workSpace, Ipiv, d_info) )
   gpuErrchkSolver( cuGETRSm(cuCholHandle, HIPBLAS_OP_N, n, nrhs, A, lda, Ipiv, B, ldb, d_info) )
#endif
   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }
}

void destroycuSolverHandle(){
   gpuErrchkSolver( hipsolverDnDestroy(cuCholHandle) )
   cuCholHandle=NULL;
}
EXTERN_C_END
